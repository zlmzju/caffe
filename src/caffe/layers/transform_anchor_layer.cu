#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include "caffe/util/gpu_util.cuh"
#include "caffe/layers/transform_anchor_layer.hpp"
using namespace std;

namespace caffe {

/*input data_matrix, output data_anchor
* input matrix T (size = 8) to projective matrix:
* T[0]+1, T[1]  , T[2]
* T[3]  , T[4]+1, T[5]
* T[6]  , T[7]  , 1
*/
template <typename Dtype>
__global__ void matrix_to_anchor(const int n, const Dtype* data_matrix,
  const int kernel_h, const int kernel_w, const Dtype stride,
  const int height_off, const int width_off,
  Dtype* data_anchor) {
  CUDA_KERNEL_LOOP(index, n) { //n is the size of (kernel_h*kernel_w, h, w)
    // index of offset and tranform matrix
    const int x_off = index % width_off;
    const int y_off = (index / width_off) % height_off;
    const int c_off = index / width_off / height_off;

    Dtype box[8] = {-1, -1, 1, -1, 1, 1, -1, 1};
    Dtype anchor[8] = {0, 0, 1, 0, 1, 1, 0, 1};
    const Dtype x_scale = (kernel_w - 1.0) / 2.0;
    const Dtype y_scale = (kernel_h - 1.0) / 2.0;
    const Dtype x_old = (box[c_off*2 + 0] * x_scale);
    const Dtype y_old = (box[c_off*2 + 1] * y_scale);
    //transform matrix multiplication: (3, 3) * (x_old, y_old, 1) = (y_new, x_new, z_new)
    Dtype T[8]; //h0, h1, ..., h7, where h8 = 1
    int idx[8]; //index for diff_matrix
    for(int i = 0; i < 8; ++i){
        idx[i] = (i * height_off + y_off) * width_off + x_off;
        T[i] = data_matrix[idx[i]];
    }

    Dtype x_new = (T[0] + 1.0) * x_old +         T[1] * y_old + T[2];
    Dtype y_new =         T[3] * x_old + (T[4] + 1.0) * y_old + T[5];
    Dtype z_new = 1.0;  //T[6] * x_old +         T[7] * y_old + 1.0;
    
    //assign new h and w to data_anchor
    int anchor_index_x = ((2 * c_off + 0) * height_off + y_off) * width_off + x_off;
    int anchor_index_y = ((2 * c_off + 1) * height_off + y_off) * width_off + x_off;
    data_anchor[anchor_index_x] = x_new / z_new - anchor[c_off*2 + 0];
    data_anchor[anchor_index_y] = y_new / z_new - anchor[c_off*2 + 1];
  }
}

template <typename Dtype>
__global__ void anchor_to_matrix(const int n, 
  const Dtype* diff_anchor, const Dtype* data_matrix,
  const int kernel_h, const int kernel_w, const Dtype stride,
  const int height_off, const int width_off,
  Dtype* diff_matrix) {
  CUDA_KERNEL_LOOP(index, n) { //n is the size of (kernel_h*kernel_w, h, w)
    // index of offset and tranform matrix
    const int x_off = index % width_off;
    const int y_off = (index / width_off) % height_off;
    const int c_off = index / width_off / height_off;

    Dtype box[8] = {-1, -1, 1, -1, 1, 1, -1, 1};
    const Dtype x_scale = (kernel_w - 1.0) / 2.0;
    const Dtype y_scale = (kernel_h - 1.0) / 2.0;
    const Dtype x_old = (box[c_off*2 + 0] * x_scale);
    const Dtype y_old = (box[c_off*2 + 1] * y_scale);
    //transform matrix multiplication: (3, 3) * (x_old, y_old, 1) = (y_new, x_new, z_new)
    Dtype T[8]; //h0, h1, ..., h7, where h8 = 1
    int idx[8]; //index for diff_matrix
    for(int i = 0; i < 8; ++i){
        idx[i] = (i * height_off + y_off) * width_off + x_off;
    }
    
    //assign new h and w to data_anchor
    int anchor_index_x = ((2 * c_off + 0) * height_off + y_off) * width_off + x_off;
    int anchor_index_y = ((2 * c_off + 1) * height_off + y_off) * width_off + x_off;
    Dtype dx = diff_anchor[anchor_index_x];
    Dtype dy = diff_anchor[anchor_index_y];

    //diff matrix values
    T[0] = (1.0 * dx * x_old); 
    T[1] = (1.0 * dx * y_old); 
    T[2] = (1.0 * dx *   1.0); 

    T[3] = (1.0 * dy * x_old); 
    T[4] = (1.0 * dy * y_old); 
    T[5] = (1.0 * dy *   1.0); 

    T[6] = 0.0; 
    T[7] = 0.0; 

    //atomic add
    for(int i = 0; i < 8; ++i){
        caffe_gpu_atomic_add(T[i], diff_matrix + idx[i]);
    }
  }
}
template <typename Dtype>
void TransformAnchorLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
    const Dtype* matrix = bottom[0]->gpu_data();
    Dtype* offset = top[0]->mutable_gpu_data();
    caffe_gpu_set(top[0]->count(), Dtype(0), offset);

    const int num_threads = top[0]->count(1) / 2;
    for(int i = 0; i < top[0]->shape(0); ++i){
        matrix_to_anchor<Dtype><<<CAFFE_GET_BLOCKS(num_threads), CAFFE_CUDA_NUM_THREADS>>>(
                num_threads, matrix + i * bottom[0]->count(1), 
                this->kernel_size, this->kernel_size, this->stride, 
                top[0]->shape(2), top[0]->shape(3), offset + i * top[0]->count(1));
    }
}

template <typename Dtype>
void TransformAnchorLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
   const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    const Dtype* matrix = bottom[0]->gpu_data();
    const Dtype* anchor_diff = top[0]->gpu_diff();
    Dtype* matrix_diff = bottom[0]->mutable_gpu_diff();
    caffe_gpu_set(bottom[0]->count(), Dtype(0), matrix_diff);

    const int num_threads = top[0]->count(1) / 2;
    for(int i = 0; i < top[0]->shape(0); ++i){
        anchor_to_matrix<Dtype><<<CAFFE_GET_BLOCKS(num_threads), CAFFE_CUDA_NUM_THREADS>>>(
                num_threads, anchor_diff + i * top[0]->count(1), matrix + i * bottom[0]->count(1),
                this->kernel_size, this->kernel_size, this->stride, 
                top[0]->shape(2), top[0]->shape(3), 
                matrix_diff + i * bottom[0]->count(1));
    }

}
//

INSTANTIATE_LAYER_GPU_FUNCS(TransformAnchorLayer);

}  // namespace caffe
